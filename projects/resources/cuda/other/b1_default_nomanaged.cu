#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdlib>
#include <ctime>
#include <math.h> 
#include <iostream>
#include "utils.hpp"
#include "options.hpp"

/////////////////////////////
/////////////////////////////

namespace chrono = std::chrono;
using clock_type = chrono::high_resolution_clock;

/////////////////////////////
/////////////////////////////

extern "C" __global__ void square(const float* x, float* y, int n) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        // float tmp = x[i];
        // float sum = 0;
        // for (int j = 0; j < 4; j++) {
        //     sum += tmp + j;
        // }

        y[i] = x[i]; // tmp + tmp * tmp / 2 + tmp * tmp * tmp / 6;
    }
}

__inline__ __device__ float warp_reduce(float val) {
    int warp_size = 32;
    for (int offset = warp_size / 2; offset > 0; offset /= 2) 
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    return val;
}

// __device__ float atomicAddDouble(float* address, float val) {
//     unsigned long long int* address_as_ull = (unsigned long long int*) address;
//     unsigned long long int old = *address_as_ull, assumed;
//     do {
//         assumed = old;
//         old = atomicCAS(address_as_ull, assumed, __float_as_longlong(val + __longlong_as_float(assumed)));
//     } while (assumed != old);
//     return __longlong_as_float(old);
// }

__global__ void reduce(const float *x, const float *y, float* z, int N) {
    int warp_size = 32;
    float sum = float(0);
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        sum += x[i] - y[i];
    }
    sum = warp_reduce(sum); // Obtain the sum of values in the current warp;
    if ((threadIdx.x & (warp_size - 1)) == 0) // Same as (threadIdx.x % warp_size) == 0 but faster
        atomicAdd(z, sum); // The first thread in the warp updates the output;
}

/////////////////////////////
/////////////////////////////

void init(float *x, float *y, int N) {
    for (int i = 0; i < N; i++) {
        x[i] = 1.0 / (i + 1);
        y[i] = 2.0 / (i + 1);
    }
}

void reset(float *res, float *res_d) {
    res[0] = 0.0;
    hipMemcpy(res_d, res, sizeof(float), hipMemcpyHostToDevice);
}

/////////////////////////////
/////////////////////////////

int main(int argc, char *argv[]) {

    srand(time(0));

    Options options = Options(argc, argv);
	int debug = options.debug;

    int num_executions = options.num_iter;
    int N = options.N;

    int block_size = options.block_size_1d;
    int num_blocks = options.num_blocks;
    int skip_iterations = options.skip_iterations;
    int err = 0;

    if (debug) {
        std::cout << "running b1 dag" << std::endl;
        std::cout << "N=" << N << std::endl;
        std::cout << "num executions=" << num_executions << std::endl;
        std::cout << "block size 1d=" << block_size << std::endl;
        std::cout << "num blocks=" << num_blocks << std::endl;
        std::cout << "skip iteration time=" << skip_iterations << std::endl;
    }
    
    auto start = clock_type::now();
    float *x_d, *y_d, *x1_d, *y1_d, *res_d, *x, *y, *res;
    x = (float*) malloc(sizeof(float) * N);
    y = (float*) malloc(sizeof(float) * N);
    res = (float*) malloc(sizeof(float));
    err = hipMalloc(&x_d, sizeof(float) * N);
    err = hipMalloc(&y_d, sizeof(float) * N);
    err = hipMalloc(&x1_d, sizeof(float) * N);
    err = hipMalloc(&y1_d, sizeof(float) * N);
    err = hipMalloc(&res_d, sizeof(float));
    if (debug && err) std::cout << err << std::endl;

    hipHostRegister(x, sizeof(float) * N, 0);
    hipHostRegister(y, sizeof(float) * N, 0);
    hipHostRegister(res, sizeof(float), 0);

    // Create streams;
    hipStream_t s1, s2;
    err = hipStreamCreate(&s1);
    err = hipStreamCreate(&s2);
    if (debug && err) std::cout << err << std::endl;

    // Initialze arrays;
    init(x, y, N);

    auto end = clock_type::now();
    if (debug) std::cout << "init=" << (float) chrono::duration_cast<chrono::microseconds>(end - start).count() / 1000 << " ms" << std::endl;

    // Print header;
    if (!debug) std::cout << "num_iter,gpu_result,total_time_sec,overhead_sec,computation_sec" << std::endl;
	
    float tot = 0;
    for (int i = 0; i < num_executions; i++) {
        if (debug) std::cout << "\n-- iter=" << i << std::endl;
        auto start_tmp = clock_type::now();
        reset(res, res_d);
        auto end_tmp = clock_type::now();
        auto reset_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
        if (debug) std::cout << "  reset=" << (float) reset_time / 1000 << " ms" << std::endl;
        
        start = clock_type::now();

        hipMemcpyAsync(x_d, x, sizeof(float) * N, hipMemcpyHostToDevice, s1);
        hipMemcpyAsync(y_d, y, sizeof(float) * N, hipMemcpyHostToDevice, s2);

        square<<<num_blocks, block_size, 0, s1>>>(x_d, x1_d, N);
        square<<<num_blocks, block_size, 0, s2>>>(y_d, y1_d, N);

        // Stream 1 waits stream 2;
        hipEvent_t e1;
        hipEventCreate(&e1);
        hipEventRecord(e1, s2);
        hipStreamWaitEvent(s1, e1, 0);

        reduce<<<num_blocks, block_size, 0, s1>>>(x1_d, y1_d, res_d, N);       
        hipMemcpyAsync(res, res_d, sizeof(float), hipMemcpyDeviceToHost, s1);
        hipStreamSynchronize(s1); 

        if (debug && err) std::cout << err << std::endl;

        end = clock_type::now();
        auto tmp = chrono::duration_cast<chrono::microseconds>(end - start).count();
        if (i >= skip_iterations) tot += tmp;

        if (debug) {
            std::cout << "  gpu result=" << res[0] << "; time=" << (float) tmp / 1000 << " ms" << std::endl;
        } else {
            std::cout << i << "," << res[0] << "," << (float) (reset_time + tmp) / 1e6 << "," << (float) reset_time / 1e6 << "," << (float) tmp / 1e6 << std::endl;
        }
    }

    // Print;
	hipDeviceSynchronize();
    
    if (debug) std::cout << "\nmean exec time=" << (float) tot / (1000 * (num_executions - skip_iterations)) << " ms" << std::endl;
}
