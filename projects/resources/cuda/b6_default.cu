#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdlib>
#include <ctime>
#include <math.h> 
#include <iostream>
#include "utils.hpp"
#include "options.hpp"

/////////////////////////////
/////////////////////////////

namespace chrono = std::chrono;
using clock_type = chrono::high_resolution_clock;

/////////////////////////////
/////////////////////////////

extern "C" __global__ void nb_1(const int* x, const float* y, float* z, int size, int n_feat, int n_classes) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < n_classes; j++) {
            for (int q = 0; q < n_feat; q++) {
                z[i * n_classes + j] += x[i * n_feat + q] * y[j * n_feat + q];
            } 
        }
    }
}

extern "C" __global__ void nb_2(const float* x, float* y, int n_row_x, int n_col_x) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        float curr_max = x[i * n_col_x];
        for (int j = 0; j < n_col_x; j++) {
            curr_max = fmaxf(curr_max, x[i * n_col_x + j]); 
        }
        y[i] = curr_max;
    }
}

extern "C" __global__ void nb_3(const float* x, const float* y, float* z, int n_row_x, int n_col_x) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        float sum = 0;
        for (int j = 0; j < n_col_x; j++) {
            sum += expf(x[i * n_col_x + j] - y[i]);
        }
        z[i] = logf(sum) + y[i];
    }
}

extern "C" __global__ void nb_4(float* x, float* y, int n_row_x, int n_col_x) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < n_col_x; j++) {
            x[i * n_col_x + j] = expf(x[i * n_col_x + j] - y[i]);
        }
    }
}

extern "C" __global__ void rr_1(const int* x, float *y, int n_row_x, int n_col_x) {
    for(int j = blockIdx.x * blockDim.x + threadIdx.x; j < n_col_x; j += blockDim.x * gridDim.x) {
        float feature_mean = 0;
        float sum_sq = 0;
        // Compute mean and variance;
        for (int i = 0; i < n_row_x; i++) {
            feature_mean += x[j * n_row_x + i];
            sum_sq += x[j * n_row_x + i] * x[j * n_row_x + i];
        }
        feature_mean /= n_row_x;
        float std = sqrtf(sum_sq / n_row_x - feature_mean * feature_mean);
        
        // Update values;
        for (int i = 0; i < n_row_x; i++) {
            y[j * n_row_x + i] = ((float) x[j * n_row_x + i] - feature_mean) / std;
        }
    }
}

extern "C" __global__ void rr_2(const float* x, const float* y, float* z, int size, int n_feat, int n_classes) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < n_classes; j++) {
            for (int q = 0; q < n_feat; q++) {
                z[i * n_classes + j] += x[i * n_feat + q] * y[j * n_feat + q];
            }
        }
    }
}

extern "C" __global__ void rr_3(float* x, const float *y, int n_row_x, int n_col_x) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < n_col_x; j++) {
            x[i * n_col_x + j] += y[j];
        }
    }
}

extern "C" __global__ void softmax(float *x, int n_row_x, int n_col_x) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        float row_exp_sum = 0;
        for (int j = 0; j < n_col_x; j++) {
            row_exp_sum += expf( x[i * n_col_x + j]);
        }
        for (int j = 0; j < n_col_x; j++) {
             x[i * n_col_x + j] = expf(x[i * n_col_x + j]) / row_exp_sum;
        }
    }
}

extern "C" __global__ void argmax(const float *x, const float *y, int *z, int n_row_x, int n_col_x) {
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        int curr_best_index = 0;
        float curr_best = x[i * n_col_x] + y[i * n_col_x];
        for (int j = 0; j < n_col_x; j++) {
            float curr = x[i * n_col_x + j] + y[i * n_col_x + j];
            if (curr > curr_best) {
                curr_best = curr;
                curr_best_index = j;
            }
        }
        z[i] = curr_best_index;
    }
}

/////////////////////////////
/////////////////////////////

void reset(float *r1, float *r2, const float *nb_class_log_prior, int N, int num_classes) {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < num_classes; j++) {
            r1[i * num_classes + j] = nb_class_log_prior[j];
            r2[i * num_classes + j] = 0;
        }
    }
}

/////////////////////////////
/////////////////////////////

int main(int argc, char *argv[]) {

    srand(time(0));

    Options options = Options(argc, argv);
	int debug = options.debug;

    int num_executions = options.num_iter;
    int N = options.N;

    int block_size = options.block_size_1d;
    int num_blocks = options.num_blocks;
    int skip_iterations = options.skip_iterations;
    int err = 0;

    int num_features = 200;
    int num_classes = 10;

    if (debug) {
        std::cout << "running b6 dag" << std::endl;
        std::cout << "N=" << N << std::endl;
        std::cout << "num executions=" << num_executions << std::endl;
        std::cout << "block size 1d=" << block_size << std::endl;
        std::cout << "num blocks=" << num_blocks << std::endl;
        std::cout << "skip iteration time=" << skip_iterations << std::endl;
    }
    
    auto start = clock_type::now();
    int *x;
    float *z;
    err = hipMallocManaged(&x, sizeof(int) * N * num_features);
    if (err) std::cout << err << std::endl;
    err = hipMallocManaged(&z, sizeof(float) * N * num_features);
    if (err) std::cout << err << std::endl;
    
    float *nb_feat_log_prob, *nb_class_log_prior, *ridge_coeff, *ridge_intercept, *nb_amax, *nb_l, *r1, *r2;
    int *r;
    err = hipMallocManaged(&nb_feat_log_prob, sizeof(float) * num_classes * num_features);
    err = hipMallocManaged(&nb_class_log_prior, sizeof(float) * num_classes);
    err = hipMallocManaged(&ridge_coeff, sizeof(float) * num_classes * num_features);
    err = hipMallocManaged(&ridge_intercept, sizeof(float) * num_classes);
    err = hipMallocManaged(&nb_amax, sizeof(float) * N);
    err = hipMallocManaged(&nb_l, sizeof(float) * N);
    err = hipMallocManaged(&r1, sizeof(float) * N * num_classes);
    err = hipMallocManaged(&r2, sizeof(float) * N * num_classes);
    err = hipMallocManaged(&r, sizeof(int) * N);
    if (err) std::cout << err << std::endl;

    // Initialze arrays;
    for (int i = 0; i < num_classes; i++) {
        for (int j = 0; j < num_features; j++) {
            nb_feat_log_prob[i * num_features + j] = (float)(rand()) / (float)(RAND_MAX);
            ridge_coeff[i * num_features + j] = (float)(rand()) / (float)(RAND_MAX);
        }
        nb_class_log_prior[i] = (float)(rand()) / (float)(RAND_MAX);
        ridge_intercept[i] = (float)(rand()) / (float)(RAND_MAX);
    }
    int max_occurrence_of_ngram = 10;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < num_features; j++) {
            x[i * num_features + j] = rand() % max_occurrence_of_ngram;
        }
        for (int j = 0; j < num_classes; j++) {
            r1[i * num_classes + j] = nb_class_log_prior[j];
            r2[i * num_classes + j] = 0;
        }
    }

    // Create streams;
    hipStream_t s1, s2;
    err = hipStreamCreate(&s1);
    err = hipStreamCreate(&s2);
    if (debug && err) std::cout << err << std::endl;

    auto end = clock_type::now();
    if (debug) std::cout << "init=" << (float) chrono::duration_cast<chrono::microseconds>(end - start).count() / 1000 << " ms" << std::endl;

    // Print header;
    if (!debug) std::cout << "num_iter,gpu_result,total_time_sec,overhead_sec,computation_sec" << std::endl;
	
    float tot = 0;
    for (int i = 0; i < num_executions; i++) {
        if (debug) std::cout << "\n-- iter=" << i << std::endl;
        auto start_tmp = clock_type::now();
        reset(r1, r2, nb_class_log_prior, N, num_classes);
        auto end_tmp = clock_type::now();
        auto reset_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
        if (debug) std::cout << " reset=" << (float) reset_time / 1000 << " ms" << std::endl;
        
        start = clock_type::now();

        hipStreamAttachMemAsync(s1, z, 0);
        hipStreamAttachMemAsync(s2, nb_feat_log_prob, 0);
        hipStreamAttachMemAsync(s2, r1, 0);
        hipStreamAttachMemAsync(s1, ridge_coeff, 0);
        hipStreamAttachMemAsync(s1, r2, 0);
        hipStreamAttachMemAsync(s2, nb_amax, 0);
        hipStreamAttachMemAsync(s2, nb_l, 0);
        hipStreamAttachMemAsync(s1, ridge_intercept, 0);

        rr_1<<<num_blocks, block_size, 0, s1>>>(x, z, N, num_features);
       
        nb_1<<<num_blocks, block_size, 0, s2>>>(x, nb_feat_log_prob, r1, N, num_features, num_classes);
       
        rr_2<<<num_blocks, block_size, 0, s1>>>(z, ridge_coeff, r2, N, num_features, num_classes);
       
        nb_2<<<num_blocks, block_size, 0, s2>>>(r1, nb_amax, N, num_classes);
       
        nb_3<<<num_blocks, block_size, 0, s2>>>(r1, nb_amax, nb_l, N, num_classes);
       
        rr_3<<<num_blocks, block_size, 0, s1>>>(r2, ridge_intercept, N, num_classes);
       
        nb_4<<<num_blocks, block_size, 0, s2>>>(r1, nb_l, N, num_classes);
       
        softmax<<<num_blocks, block_size, 0, s2>>>(r1, N, num_classes);
        
        softmax<<<num_blocks, block_size, 0, s1>>>(r2, N, num_classes);
        
        // Stream 1 waits stream 2;
        hipEvent_t e1;
        hipEventCreate(&e1);
        hipEventRecord(e1, s2);
        hipStreamWaitEvent(s1, e1, 0);

        argmax<<<num_blocks, block_size, 0, s1>>>(r1, r2, r, N, num_classes);
        hipDeviceSynchronize();
       
        end = clock_type::now();
        auto tmp = chrono::duration_cast<chrono::microseconds>(end - start).count();
        if (i >= skip_iterations) tot += tmp;

        if (debug) {
            std::cout << "  gpu result=[";
            for (int j = 0; j < 10; j++) {
                std::cout << r[j] << ", ";
            } 
            std::cout << ", ...]; time=" << (float) tmp / 1000 << " ms" << std::endl;
        } else {
            std::cout << i << "," << 0.0 << "," << (float) (reset_time + tmp) / 1e6 << "," << (float) reset_time / 1e6 << "," << (float) tmp / 1e6 << std::endl;
        }
    }

    // Print;
	hipDeviceSynchronize();
    
    if (debug) std::cout << "\nmean exec time=" << (float) tot / (1000 * (num_executions - skip_iterations)) << " ms" << std::endl;
}
