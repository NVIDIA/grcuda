#include "hip/hip_runtime.h"
#include "b7.cuh"

//////////////////////////////
//////////////////////////////

#define WARP_SIZE 32
#define THREADS_PER_VECTOR 4
#define MAX_NUM_VECTORS_PER_BLOCK (1024 / THREADS_PER_VECTOR)

/////////////////////////////
/////////////////////////////

extern "C" __global__ void spmv(const int *ptr, const int *idx, const int *val, const float *vec, float *res, int num_rows, int num_nnz) {
    for (int n = blockIdx.x * blockDim.x + threadIdx.x; n < num_rows; n += blockDim.x * gridDim.x) {
        float sum = 0;
        for (int i = ptr[n]; i < ptr[n + 1]; i++) {
            sum += val[i] * vec[idx[i]];
        }
        res[n] = sum;
    }
}

extern "C" __global__ void spmv2(const int *ptr, const int *idx, const int *val, const float *vec, float *res, int num_rows, int num_nnz) {
    // Thread ID in block
    int t = threadIdx.x;

    // Thread ID in warp
    int lane = t & (WARP_SIZE - 1);

    // Number of warps per block
    int warpsPerBlock = blockDim.x / WARP_SIZE;

    // One row per warp
    int row = (blockIdx.x * warpsPerBlock) + (t / WARP_SIZE);

    extern __shared__ volatile float vals[];

    if (row < num_rows) {
        int rowStart = ptr[row];
        int rowEnd = ptr[row + 1];
        float sum = 0;

        // Use all threads in a warp accumulate multiplied elements
        for (int j = rowStart + lane; j < rowEnd; j += WARP_SIZE) {
            int col = idx[j];
            sum += val[j] * vec[col];
        }
        vals[t] = sum;
        __syncthreads();

        // Reduce partial sums
        if (lane < 16) vals[t] += vals[t + 16];
        if (lane < 8) vals[t] += vals[t + 8];
        if (lane < 4) vals[t] += vals[t + 4];
        if (lane < 2) vals[t] += vals[t + 2];
        if (lane < 1) vals[t] += vals[t + 1];
        __syncthreads();

        // Write result
        if (lane == 0) {
            res[row] = vals[t];
        }
    }
}

extern "C" __global__ void spmv3(int *cudaRowCounter, int *d_ptr, int *d_cols, int *d_val, float *d_vector, float *d_out, int N) {
    int i;
    float sum;
    int row;
    int rowStart, rowEnd;
    int laneId = threadIdx.x % THREADS_PER_VECTOR;       //lane index in the vector
    int vectorId = threadIdx.x / THREADS_PER_VECTOR;     //vector index in the thread block
    int warpLaneId = threadIdx.x & 31;                   //lane index in the warp
    int warpVectorId = warpLaneId / THREADS_PER_VECTOR;  //vector index in the warp

    __shared__ volatile int space[MAX_NUM_VECTORS_PER_BLOCK][2];

    // Get the row index
    if (warpLaneId == 0) {
        row = atomicAdd(cudaRowCounter, 32 / THREADS_PER_VECTOR);
    }
    // Broadcast the value to other threads in the same warp and compute the row index of each vector
    row = __shfl_sync(0xffffffff, row, 0) + warpVectorId;

    while (row < N) {
        // Use two threads to fetch the row offset
        if (laneId < 2) {
            space[vectorId][laneId] = d_ptr[row + laneId];
        }
        rowStart = space[vectorId][0];
        rowEnd = space[vectorId][1];

        sum = 0;
        // Compute dot product
        if (THREADS_PER_VECTOR == 32) {
            // Ensure aligned memory access
            i = rowStart - (rowStart & (THREADS_PER_VECTOR - 1)) + laneId;

            // Process the unaligned part
            if (i >= rowStart && i < rowEnd) {
                sum += d_val[i] * d_vector[d_cols[i]];
            }

            // Process the aligned part
            for (i += THREADS_PER_VECTOR; i < rowEnd; i += THREADS_PER_VECTOR) {
                sum += d_val[i] * d_vector[d_cols[i]];
            }
        } else {
            for (i = rowStart + laneId; i < rowEnd; i += THREADS_PER_VECTOR) {
                sum += d_val[i] * d_vector[d_cols[i]];
            }
        }
        // Intra-vector reduction
        for (i = THREADS_PER_VECTOR >> 1; i > 0; i >>= 1) {
            sum += __shfl_down_sync(0xffffffff, sum, i);
        }

        // Save the results
        if (laneId == 0) {
            d_out[row] = sum;
        }

        // Get a new row index
        if (warpLaneId == 0) {
            row = atomicAdd(cudaRowCounter, 32 / THREADS_PER_VECTOR);
        }
        // Broadcast the row index to the other threads in the same warp and compute the row index of each vector
        row = __shfl_sync(0xffffffff, row, 0) + warpVectorId;
    }
}

__inline__ __device__ float warp_reduce(float val) {
    int warp_size = 32;
    for (int offset = warp_size / 2; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    return val;
}

extern "C" __global__ void sum(const float *x, float *z, int N) {
    int warp_size = 32;
    float sum = float(0);
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        sum += x[i];
    }
    sum = warp_reduce(sum);                    // Obtain the sum of values in the current warp;
    if ((threadIdx.x & (warp_size - 1)) == 0)  // Same as (threadIdx.x % warp_size) == 0 but faster
        atomicAdd(z, sum);                     // The first thread in the warp updates the output;
}

extern "C" __global__ void divide(const float *x, float *y, float *val, int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        y[i] = x[i] / val[0];
    }
}

extern "C" __global__ void reset_kernel(float *n1, float *n2, int *r1, int *r2) {
    if (blockIdx.x * blockDim.x + threadIdx.x == 0) {
        *n1 = 0;
        *n2 = 0;
        *r1 = 0;
        *r2 = 0;
    }
}

//////////////////////////////
//////////////////////////////

void CUDART_CB host_callback(void *data) {
    // Check status of GPU after stream operations are done
    callBackData_t *tmp = (callBackData_t *)(data);
    tmp->n1[0] = 0.0;
    tmp->n2[0] = 0.0;
    tmp->r1[0] = 0;
    tmp->r2[0] = 0;
}

//////////////////////////////
//////////////////////////////

void Benchmark7::alloc() {
    nnz = degree * N;
    ptr_tmp = (int *)malloc(sizeof(int) * (N + 1));
    ptr2_tmp = (int *)malloc(sizeof(int) * (N + 1));
    idx_tmp = (int *)malloc(sizeof(int) * nnz);
    idx2_tmp = (int *)malloc(sizeof(int) * nnz);
    val_tmp = (int *)malloc(sizeof(int) * nnz);
    val2_tmp = (int *)malloc(sizeof(int) * nnz);

    err = hipMallocManaged(&ptr, sizeof(int) * (N + 1));
    err = hipMallocManaged(&ptr2, sizeof(int) * (N + 1));
    err = hipMallocManaged(&idx, sizeof(int) * nnz);
    err = hipMallocManaged(&idx2, sizeof(int) * nnz);
    err = hipMallocManaged(&val, sizeof(int) * nnz);
    err = hipMallocManaged(&val2, sizeof(int) * nnz);
    err = hipMallocManaged(&rowCounter1, sizeof(int));
    err = hipMallocManaged(&rowCounter2, sizeof(int));

    err = hipMallocManaged(&auth1, sizeof(float) * N);
    err = hipMallocManaged(&auth2, sizeof(float) * N);
    err = hipMallocManaged(&hub1, sizeof(float) * N);
    err = hipMallocManaged(&hub2, sizeof(float) * N);
    err = hipMallocManaged(&auth_norm, sizeof(float));
    err = hipMallocManaged(&hub_norm, sizeof(float));

    x = (int *)malloc(nnz * sizeof(int));
    y = (int *)malloc(nnz * sizeof(int));
    v = (int *)malloc(nnz * sizeof(int));

    err = hipStreamCreate(&s1);
    err = hipStreamCreate(&s2);
}

void Benchmark7::init() {
    random_coo(x, y, v, N, degree);
    // Create a CSR;
    coo2csr(ptr_tmp, idx_tmp, val_tmp, x, y, v, N, N, nnz);
    coo2csr(ptr2_tmp, idx2_tmp, val2_tmp, y, x, v, N, N, nnz);
}

void Benchmark7::reset() {
    for (int j = 0; j < nnz; j++) {
        idx[j] = idx_tmp[j];
        idx2[j] = idx2_tmp[j];
        val[j] = val_tmp[j];
        val2[j] = val2_tmp[j];
    }
    for (int j = 0; j < N + 1; j++) {
        ptr[j] = ptr_tmp[j];
        ptr2[j] = ptr2_tmp[j];
    }
    for (int i = 0; i < N; i++) {
        auth1[i] = 1;
        auth2[i] = 1;
        hub1[i] = 1;
        hub2[i] = 1;
    }
    auth_norm[0] = 0;
    hub_norm[0] = 0;
    rowCounter1[0] = 0;
    rowCounter2[0] = 0;
}

void Benchmark7::execute_sync(int iter) {
    for (int iter = 0; iter < iterations; iter++) {
        // hipMemPrefetchAsync(auth1, N * sizeof(float), 0);
        // hipMemPrefetchAsync(auth2, N * sizeof(float), 0);
        // hipMemPrefetchAsync(hub1, N * sizeof(float), 0);
        // hipMemPrefetchAsync(hub2, N * sizeof(float), 0);
        // hipMemPrefetchAsync(auth_norm, sizeof(float), 0);
        // hipMemPrefetchAsync(hub_norm, sizeof(float), 0);
        // hipDeviceSynchronize();

        int nb = ceil(N / ((float)block_size_1d));

        // spmv<<<nb, block_size_1d>>>(ptr2, idx2, val2, hub1, auth2, N, nnz);
        spmv3<<<nb, block_size_1d, block_size_1d * sizeof(float)>>>(rowCounter1, ptr2, idx2, val2, hub1, auth2, N);
        err = hipDeviceSynchronize();

        // spmv<<<nb, block_size_1d>>>(ptr, idx, val, auth1, hub2, N, nnz);
        spmv3<<<nb, block_size_1d, block_size_1d * sizeof(float)>>>(rowCounter2, ptr, idx, val, auth1, hub2, N);
        err = hipDeviceSynchronize();

        sum<<<num_blocks, block_size_1d>>>(auth2, auth_norm, N);
        err = hipDeviceSynchronize();

        sum<<<num_blocks, block_size_1d>>>(hub2, hub_norm, N);
        err = hipDeviceSynchronize();

        divide<<<num_blocks, block_size_1d>>>(auth2, auth1, auth_norm, N);
        err = hipDeviceSynchronize();

        divide<<<num_blocks, block_size_1d>>>(hub2, hub1, hub_norm, N);
        err = hipDeviceSynchronize();

        auth_norm[0] = 0;
        hub_norm[0] = 0;
        rowCounter1[0] = 0;
        rowCounter2[0] = 0;

        if (debug && err) std::cout << err << std::endl;
    }
}

void Benchmark7::execute_async(int iter) {
    for (int iter = 0; iter < iterations; iter++) {
        // hipMemPrefetchAsync(auth1, N * sizeof(float), 0, s2);
        // hipMemPrefetchAsync(auth2, N * sizeof(float), 0, s1);
        // hipMemPrefetchAsync(hub1, N * sizeof(float), 0, s1);
        // hipMemPrefetchAsync(hub2, N * sizeof(float), 0, s2);
        // hipMemPrefetchAsync(auth_norm, sizeof(float), 0, s1);
        // hipMemPrefetchAsync(hub_norm, sizeof(float), 0, s2);

        hipStreamAttachMemAsync(s1, ptr2, 0);
        hipStreamAttachMemAsync(s1, idx2, 0);
        hipStreamAttachMemAsync(s1, val2, 0);
        hipStreamAttachMemAsync(s1, hub1, 0);
        hipStreamAttachMemAsync(s1, auth2, 0);

        hipStreamAttachMemAsync(s2, ptr, 0);
        hipStreamAttachMemAsync(s2, idx, 0);
        hipStreamAttachMemAsync(s2, val, 0);
        hipStreamAttachMemAsync(s2, auth1, 0);
        hipStreamAttachMemAsync(s2, hub2, 0);

        hipEvent_t e1, e2;
        hipEventCreate(&e1);
        hipEventCreate(&e2);

        int nb = ceil(N / ((float)block_size_1d));

        // spmv<<<nb, block_size_1d, 0, s1>>>(ptr2, idx2, val2, hub1, auth2, N, nnz);
        spmv3<<<nb, block_size_1d, block_size_1d * sizeof(float), s1>>>(rowCounter1, ptr2, idx2, val2, hub1, auth2, N);
        err = hipEventRecord(e1, s1);
        // spmv<<<nb, block_size_1d, 0, s2>>>(ptr, idx, val, auth1, hub2, N, nnz);
        spmv3<<<nb, block_size_1d, block_size_1d * sizeof(float), s2>>>(rowCounter2, ptr, idx, val, auth1, hub2, N);
        err = hipEventRecord(e2, s2);

        sum<<<num_blocks, block_size_1d, 0, s1>>>(auth2, auth_norm, N);

        sum<<<num_blocks, block_size_1d, 0, s2>>>(hub2, hub_norm, N);

        // Stream 1 waits stream 2;
        err = hipStreamWaitEvent(s1, e2, 0);
        hipStreamAttachMemAsync(s1, auth1, 0);
        divide<<<num_blocks, block_size_1d, 0, s1>>>(auth2, auth1, auth_norm, N);
        // Stream 2 waits stream 1;
        err = hipStreamWaitEvent(s2, e1, 0);
        hipStreamAttachMemAsync(s2, hub1, 0);
        divide<<<num_blocks, block_size_1d, 0, s2>>>(hub2, hub1, hub_norm, N);

        // hipEvent_t e3;
        // hipEventCreate(&e3);
        // hipEventRecord(e3, s2);
        // checkCudaErrors(hipStreamWaitEvent(s1, e3, 0));
        // reset_kernel<<<1, 1, 0, s1>>>(auth_norm, hub_norm, rowCounter1, rowCounter2);

        err = hipStreamSynchronize(s1);
        err = hipStreamSynchronize(s2);
        auth_norm[0] = 0;
        hub_norm[0] = 0;
        rowCounter1[0] = 0;
        rowCounter2[0] = 0;

        if (debug && err) std::cout << err << std::endl;
    }
    // err = hipStreamSynchronize(s1);
}

void Benchmark7::execute_cudagraph(int iter) {
    if (iter == 0) {
        hipEvent_t ef;
        hipEventCreate(&ef);
        hipStreamBeginCapture(s1, hipStreamCaptureModeGlobal);
        hipEventRecord(ef, s1);
        hipStreamWaitEvent(s2, ef, 0);

        // callBackData_t hostFnData = {auth_norm, hub_norm, rowCounter1, rowCounter2};
        // hipHostFn_t fn = host_callback;

        for (int i = 0; i < iterations; i++) {
            hipEvent_t e1, e2;
            hipEventCreate(&e1);
            hipEventCreate(&e2);

            int nb = ceil(N / ((float)block_size_1d));

            // spmv<<<nb, block_size_1d, 0, s1>>>(ptr2, idx2, val2, hub1, auth2, N, nnz);
            spmv3<<<nb, block_size_1d, block_size_1d * sizeof(float), s1>>>(rowCounter1, ptr2, idx2, val2, hub1, auth2, N);

            // spmv<<<nb, block_size_1d, 0, s2>>>(ptr, idx, val, auth1, hub2, N, nnz);
            spmv3<<<nb, block_size_1d, block_size_1d * sizeof(float), s2>>>(rowCounter2, ptr, idx, val, auth1, hub2, N);

            sum<<<num_blocks, block_size_1d, 0, s1>>>(auth2, auth_norm, N);
            err = hipEventRecord(e1, s1);
            sum<<<num_blocks, block_size_1d, 0, s2>>>(hub2, hub_norm, N);
            err = hipEventRecord(e2, s2);
            // Stream 1 waits stream 2;
            err = hipStreamWaitEvent(s1, e2, 0);
            divide<<<num_blocks, block_size_1d, 0, s1>>>(auth2, auth1, auth_norm, N);
            // Stream 2 waits stream 1;
            err = hipStreamWaitEvent(s2, e1, 0);
            divide<<<num_blocks, block_size_1d, 0, s2>>>(hub2, hub1, hub_norm, N);
            // Stream 1 waits stream 2;
            hipEvent_t e3;
            hipEventCreate(&e3);
            hipEventRecord(e3, s2);
            checkCudaErrors(hipStreamWaitEvent(s1, e3, 0));

            // This doesn't work for some reason;
            // checkCudaErrors(hipLaunchHostFunc(s1, fn, &hostFnData));

            reset_kernel<<<1, 1, 0, s1>>>(auth_norm, hub_norm, rowCounter1, rowCounter2);
            hipEvent_t e4;
            hipEventCreate(&e4);
            hipEventRecord(e4, s1);
            checkCudaErrors(hipStreamWaitEvent(s2, e4, 0));

        }

        checkCudaErrors(hipStreamEndCapture(s1, &graph));
        checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));
    }
    checkCudaErrors(hipGraphLaunch(graphExec, s1));
    err = hipStreamSynchronize(s1);
}

void Benchmark7::execute_cudagraph_manual(int iter) {}

std::string Benchmark7::print_result(bool short_form) {
    if (short_form) {
        return std::to_string(auth1[0]);
    } else {
        std::string res = "[";
        for (int j = 0; j < 10; j++) {
            res += std::to_string(auth1[j]) + ", ";
        }
        return res + ", ...]";
    }
}
