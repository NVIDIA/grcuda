#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdlib>
#include <ctime>
#include <math.h> 
#include <iostream>
#include "utils.hpp"
#include "options.hpp"

/////////////////////////////
/////////////////////////////

namespace chrono = std::chrono;
using clock_type = chrono::high_resolution_clock;

/////////////////////////////
/////////////////////////////

// float R = 0.08;
// float V = 0.3;
// float T = 1.0;
// float K = 60.0;

// __device__ inline float cndGPU(float d) {
//     const float       A1 = 0.31938153f;
//     const float       A2 = -0.356563782f;
//     const float       A3 = 1.781477937f;
//     const float       A4 = -1.821255978f;
//     const float       A5 = 1.330274429f;
//     const float RSQRT2PI = 0.39894228040143267793994605993438f;

//     float
//     K = __fdividef(1.0f, (1.0f + 0.2316419f * fabsf(d)));

//     float
//     cnd = RSQRT2PI * __expf(- 0.5f * d * d) *
//           (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

//     if (d > 0)
//         cnd = 1.0f - cnd;

//     return cnd;
// }

// extern "C" __global__ void bs(const float *x, float *y, int N, float R, float V, float T, float K) {

//     float sqrtT = __fdividef(1.0F, rsqrtf(T));
//     for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
//         float expRT;
//         float d1, d2, CNDD1, CNDD2;
//         d1 = __fdividef(__logf(x[i] / K) + (R + 0.5f * V * V) * T, V * sqrtT);
//         d2 = d1 - V * sqrtT;

//         CNDD1 = cndGPU(d1);
//         CNDD2 = cndGPU(d2);

//         //Calculate Call and Put simultaneously
//         expRT = __expf(-R * T);
//         y[i] = x[i] * CNDD1 - K * expRT * CNDD2;
//     }
// }

double R = 0.08;
double V = 0.3;
double T = 1.0;
double K = 60.0;

__device__ inline double cndGPU(double d) {
    const double       A1 = 0.31938153f;
    const double       A2 = -0.356563782f;
    const double       A3 = 1.781477937f;
    const double       A4 = -1.821255978f;
    const double       A5 = 1.330274429f;
    const double RSQRT2PI = 0.39894228040143267793994605993438f;

    double
    K = 1.0 / (1.0 + 0.2316419 * fabs(d));

    double
    cnd = RSQRT2PI * exp(- 0.5f * d * d) *
          (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

    if (d > 0)
        cnd = 1.0 - cnd;

    return cnd;
}

extern "C" __global__ void bs(const double *x, double *y, int N, double R, double V, double T, double K) {

    double sqrtT = 1.0 / rsqrt(T);
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        double expRT;
        double d1, d2, CNDD1, CNDD2;
        d1 = (log(x[i] / K) + (R + 0.5 * V * V) * T) / (V * sqrtT);
        d2 = d1 - V * sqrtT;

        CNDD1 = cndGPU(d1);
        CNDD2 = cndGPU(d2);

        //Calculate Call and Put simultaneously
        expRT = exp(-R * T);
        y[i] = x[i] * CNDD1 - K * expRT * CNDD2;
    }
}

/////////////////////////////
/////////////////////////////

void init(double **x, double **y, double* tmp_x, int N, int K) {
    for (int j = 0; j < N; j++) {
        tmp_x[j] = 60 - 0.5 + (double) rand() / RAND_MAX;
        for (int i = 0; i < K; i++) {
            x[i][j] = tmp_x[j];
            // y[i][j] = 0;
        }
    }
}

void reset(double **x, double* y, int N, int K, hipStream_t *s) {
    for (int i = 0; i < K; i++) {
        // memcpy(x[i], y, sizeof(int) * N);
        // hipMemcpy(x[i], y, sizeof(double) * N, hipMemcpyDefault);
        
        // hipMemcpyAsync(x[i], y, sizeof(int) * N, hipMemcpyHostToDevice, s[i]);
        for (int j = 0; j < N; j++) {
            x[i][j] = y[j];
        }
    }
    // hipMemPrefetchAsync(x[0], sizeof(double) * N, 0, s[0]);
}


/////////////////////////////
/////////////////////////////

int main(int argc, char *argv[]) {

    srand(time(0));

    Options options = Options(argc, argv);
	int debug = options.debug;

    int num_executions = options.num_iter;
    int N = options.N;

    int block_size = options.block_size_1d;
    int num_blocks = options.num_blocks;
    int skip_iterations = options.skip_iterations;
    int err = 0;

    int M = 10;

    if (debug) {
        std::cout << "running b5 default" << std::endl;
        std::cout << "N=" << N << std::endl;
        std::cout << "num executions=" << num_executions << std::endl;
        std::cout << "block size 1d=" << block_size << std::endl;
        std::cout << "num blocks=" << num_blocks << std::endl;
        std::cout << "skip iteration time=" << skip_iterations << std::endl;
    }
    
    auto start = clock_type::now();
    double **x = (double **) malloc(sizeof(double*) * M);
    double **y = (double **) malloc(sizeof(double*) * M);
    double *tmp_x = (double *) malloc(sizeof(double) * N);
    // hipHostRegister(tmp_x, sizeof(double) * N, 0);

    for (int i = 0; i < M; i++) {
        hipMallocManaged(&x[i], sizeof(double) * N);
        hipMallocManaged(&y[i], sizeof(double) * N);
    }
    if (debug && err) std::cout << err << std::endl;
    
    // Create streams;
    hipStream_t *s = (hipStream_t *) malloc(sizeof(hipStream_t) * M);
    for (int i = 0; i < M; i++) {
        err = hipStreamCreate(&s[i]);
    }
    if (debug && err) std::cout << err << std::endl;

    // Initialze arrays;
    init(x, y, tmp_x, N, M);

    if (debug) std::cout << "x[0][0]=" << tmp_x[0] << std::endl;

    auto end = clock_type::now();
    if (debug) std::cout << "init=" << (double) chrono::duration_cast<chrono::microseconds>(end - start).count() / 1000 << " ms" << std::endl;

    // Print header;
    if (!debug) std::cout << "num_iter,gpu_result,total_time_sec,overhead_sec,computation_sec" << std::endl;
	
    double tot = 0;
    for (int i = 0; i < num_executions; i++) {
        if (debug) std::cout << "\n-- iter=" << i << std::endl;
        auto start_tmp = clock_type::now();
        reset(x, tmp_x, N, M, s);
        auto end_tmp = clock_type::now();
        auto reset_time = chrono::duration_cast<chrono::microseconds>(end_tmp - start_tmp).count();
        if (debug) std::cout << "  reset=" << (double) reset_time / 1000 << " ms" << std::endl;
        
        start = clock_type::now();

        for (int j = 0; j < M; j++) {
            hipStreamAttachMemAsync(s[j], x[j], sizeof(double) * N);
            hipStreamAttachMemAsync(s[j], y[j], sizeof(double) * N);
            // if (j > 0) hipMemPrefetchAsync(y[j - 1], sizeof(double) * N, hipCpuDeviceId, s[j - 1]);
            bs<<<num_blocks, block_size, 0, s[j]>>>(x[j], y[j], N, R, V, T, K);
            // if (j < M - 1) hipMemPrefetchAsync(x[j + 1], sizeof(double) * N, 0, s[j + 1]);
        }

        // Last tile;
        // hipMemPrefetchAsync(y[M - 1], sizeof(double) * N, hipCpuDeviceId, s[M - 1]);

        for (int j = 0; j < M; j++) {
            err = hipStreamSynchronize(s[j]);
        }

        if (debug && err) std::cout << err << std::endl;

        end = clock_type::now();
        auto tmp = chrono::duration_cast<chrono::microseconds>(end - start).count();
        if (i >= skip_iterations) tot += tmp;

        if (debug) {
            std::cout << "  gpu result=[";
            for (int j = 0; j < M; j++) {
                std::cout << y[j][0] << ", ";
            } 
            std::cout << ", ...]; time=" << (double) tmp / 1000 << " ms" << std::endl;
        } else {
            std::cout << i << "," << y[0][0] << "," << (double) (reset_time + tmp) / 1e6 << "," << (double) reset_time / 1e6 << "," << (double) tmp / 1e6 << std::endl;
        }
    }

    // Print;
	hipDeviceSynchronize();
    
    if (debug) std::cout << "\nmean exec time=" << (double) tot / (1000 * (num_executions - skip_iterations)) << " ms" << std::endl;
}
