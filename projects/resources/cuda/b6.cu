#include "hip/hip_runtime.h"
#include "b6.cuh"

//////////////////////////////
//////////////////////////////

extern "C" __global__ void nb_1(const int* x, const float* y, float* z, int size, int n_feat, int n_classes) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < n_classes; j++) {
            for (int q = 0; q < n_feat; q++) {
                z[i * n_classes + j] += x[i * n_feat + q] * y[j * n_feat + q];
            }
        }
    }
}

extern "C" __global__ void nb_2(const float* x, float* y, int n_row_x, int n_col_x) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        float curr_max = x[i * n_col_x];
        for (int j = 0; j < n_col_x; j++) {
            curr_max = fmaxf(curr_max, x[i * n_col_x + j]);
        }
        y[i] = curr_max;
    }
}

extern "C" __global__ void nb_3(const float* x, const float* y, float* z, int n_row_x, int n_col_x) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        float sum = 0;
        for (int j = 0; j < n_col_x; j++) {
            sum += expf(x[i * n_col_x + j] - y[i]);
        }
        z[i] = logf(sum) + y[i];
    }
}

extern "C" __global__ void nb_4(float* x, float* y, int n_row_x, int n_col_x) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < n_col_x; j++) {
            x[i * n_col_x + j] = expf(x[i * n_col_x + j] - y[i]);
        }
    }
}

__inline__ __device__ float warp_reduce(float val) {
    int warp_size = 32;
    for (int offset = warp_size / 2; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    return val;
}

extern "C" __global__ void rr_1_0(const int* x, float* y, float* z, int n_row_x, int n_col_x) {
    int warp_size = 32;
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < n_col_x; j += blockDim.x * gridDim.x) {
        // Compute mean and variance;
        float feature_mean = float(0);
        float sum_sq = float(0);
        for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < n_row_x; i += blockDim.y * gridDim.y) {
            float x_tmp = x[j * n_row_x + i];
            feature_mean += x_tmp;
            sum_sq += x_tmp * x_tmp;
        }
        feature_mean = warp_reduce(feature_mean);  // Obtain the sum of values in the current warp;
        sum_sq = warp_reduce(sum_sq);              // Obtain the sum of values in the current warp;
        if (!(threadIdx.y % warp_size)) {
            atomicAdd(y + j, feature_mean);
            atomicAdd(z + j, sum_sq);
        }
    }
}

extern "C" __global__ void rr_1_1(const int* x, float* y, const float* mean, const float* std, int n_row_x, int n_col_x) {
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < n_col_x; j += blockDim.x * gridDim.x) {
        float mean_tmp = mean[j] / n_row_x;
        float std_tmp = sqrtf(std[j] / n_row_x - mean_tmp * mean_tmp);

        for (int i = blockIdx.y * blockDim.y + threadIdx.y; i < n_row_x; i += blockDim.y * gridDim.y) {
            y[j * n_row_x + i] = ((float)x[j * n_row_x + i] - mean_tmp) / std_tmp;
        }
    }
}

extern "C" __global__ void rr_1(const int* x, float* y, int n_row_x, int n_col_x) {
    for (int j = blockIdx.x * blockDim.x + threadIdx.x; j < n_col_x; j += blockDim.x * gridDim.x) {
        float feature_mean = 0;
        float sum_sq = 0;
        // Compute mean and variance;
        for (int i = 0; i < n_row_x; i++) {
            float x_tmp = x[j * n_row_x + i];
            feature_mean += x_tmp;
            sum_sq += x_tmp * x_tmp;
        }
        feature_mean /= n_row_x;
        float std = sqrtf(sum_sq / n_row_x - feature_mean * feature_mean);

        // Update values;
        for (int i = 0; i < n_row_x; i++) {
            y[j * n_row_x + i] = (x[j * n_row_x + i] - feature_mean) / std;
        }
    }
}

extern "C" __global__ void rr_2(const float* x, const float* y, float* z, int size, int n_feat, int n_classes) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < n_classes; j++) {
            for (int q = 0; q < n_feat; q++) {
                z[i * n_classes + j] += x[i * n_feat + q] * y[j * n_feat + q];
            }
        }
    }
}

extern "C" __global__ void rr_3(float* x, const float* y, int n_row_x, int n_col_x) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        for (int j = 0; j < n_col_x; j++) {
            x[i * n_col_x + j] += y[j];
        }
    }
}

extern "C" __global__ void softmax(float* x, int n_row_x, int n_col_x) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        float row_exp_sum = 0;
        for (int j = 0; j < n_col_x; j++) {
            row_exp_sum += expf(x[i * n_col_x + j]);
        }
        for (int j = 0; j < n_col_x; j++) {
            x[i * n_col_x + j] = expf(x[i * n_col_x + j]) / row_exp_sum;
        }
    }
}

extern "C" __global__ void argmax(const float* x, const float* y, int* z, int n_row_x, int n_col_x) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n_row_x; i += blockDim.x * gridDim.x) {
        int curr_best_index = 0;
        float curr_best = x[i * n_col_x] + y[i * n_col_x];
        for (int j = 0; j < n_col_x; j++) {
            float curr = x[i * n_col_x + j] + y[i * n_col_x + j];
            if (curr > curr_best) {
                curr_best = curr;
                curr_best_index = j;
            }
        }
        z[i] = curr_best_index;
    }
}

//////////////////////////////
//////////////////////////////

void Benchmark6::alloc() {
    err = hipMallocManaged(&x, sizeof(int) * N * num_features);
    err = hipMallocManaged(&z, sizeof(float) * N * num_features);
    err = hipMallocManaged(&nb_feat_log_prob, sizeof(float) * num_classes * num_features);
    err = hipMallocManaged(&nb_class_log_prior, sizeof(float) * num_classes);
    err = hipMallocManaged(&ridge_coeff, sizeof(float) * num_classes * num_features);
    err = hipMallocManaged(&ridge_intercept, sizeof(float) * num_classes);
    err = hipMallocManaged(&nb_amax, sizeof(float) * N);
    err = hipMallocManaged(&nb_l, sizeof(float) * N);
    err = hipMallocManaged(&r1, sizeof(float) * N * num_classes);
    err = hipMallocManaged(&r2, sizeof(float) * N * num_classes);
    err = hipMallocManaged(&r, sizeof(int) * N);

    err = hipStreamCreate(&s1);
    err = hipStreamCreate(&s2);
}

void Benchmark6::init() {
    for (int i = 0; i < num_classes; i++) {
        for (int j = 0; j < num_features; j++) {
            nb_feat_log_prob[i * num_features + j] = (float)(rand()) / (float)(RAND_MAX);
            ridge_coeff[i * num_features + j] = (float)(rand()) / (float)(RAND_MAX);
        }
        nb_class_log_prior[i] = (float)(rand()) / (float)(RAND_MAX);
        ridge_intercept[i] = (float)(rand()) / (float)(RAND_MAX);
    }
    int max_occurrence_of_ngram = 10;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < num_features; j++) {
            x[i * num_features + j] = rand() % max_occurrence_of_ngram;
        }
        for (int j = 0; j < num_classes; j++) {
            r1[i * num_classes + j] = nb_class_log_prior[j];
            r2[i * num_classes + j] = 0;
        }
    }
}

void Benchmark6::reset() {
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < num_classes; j++) {
            r1[i * num_classes + j] = nb_class_log_prior[j];
            r2[i * num_classes + j] = 0;
        }
        // r1_mean[i] = 0;
        // r1_std[i] = 0;
    }
}

void Benchmark6::execute_sync(int iter) {
    rr_1<<<num_blocks, block_size_1d>>>(x, z, N, num_features);
    // dim3 num_blocks_2d(8, 8);
    // dim3 block_size_1d_2d(1, 32);
    // rr_1_0<<<num_blocks_2d, block_size_1d_2d>>>(x, r1_mean, r1_std, N, num_features);
    // hipDeviceSynchronize();
    // rr_1_1<<<num_blocks_2d, block_size_1d_2d>>>(x, z, r1_mean, r1_std, N, num_features);
    hipDeviceSynchronize();

    // auto e1 = clock_type::now();
    // auto rr1time = chrono::duration_cast<chrono::microseconds>(e1 - start).count();
    // if (debug) std::cout << " rr1=" << (float) rr1time / 1000 << " ms" << std::endl;

    nb_1<<<num_blocks, block_size_1d>>>(x, nb_feat_log_prob, r1, N, num_features, num_classes);
    hipDeviceSynchronize();

    rr_2<<<num_blocks, block_size_1d>>>(z, ridge_coeff, r2, N, num_features, num_classes);
    hipDeviceSynchronize();

    nb_2<<<num_blocks, block_size_1d>>>(r1, nb_amax, N, num_classes);
    hipDeviceSynchronize();

    nb_3<<<num_blocks, block_size_1d>>>(r1, nb_amax, nb_l, N, num_classes);
    hipDeviceSynchronize();

    rr_3<<<num_blocks, block_size_1d>>>(r2, ridge_intercept, N, num_classes);
    hipDeviceSynchronize();

    nb_4<<<num_blocks, block_size_1d>>>(r1, nb_l, N, num_classes);
    hipDeviceSynchronize();

    softmax<<<num_blocks, block_size_1d>>>(r1, N, num_classes);
    hipDeviceSynchronize();

    softmax<<<num_blocks, block_size_1d>>>(r2, N, num_classes);
    hipDeviceSynchronize();

    argmax<<<num_blocks, block_size_1d>>>(r1, r2, r, N, num_classes);
    hipDeviceSynchronize();
}

void Benchmark6::execute_async(int iter) {
    hipStreamAttachMemAsync(s1, z, 0);
    // hipStreamAttachMemAsync(s1, r1_mean, 0);
    // hipStreamAttachMemAsync(s1, r1_std, 0);
    hipStreamAttachMemAsync(s2, nb_feat_log_prob, 0);
    hipStreamAttachMemAsync(s2, r1, 0);
    hipStreamAttachMemAsync(s1, ridge_coeff, 0);
    hipStreamAttachMemAsync(s1, r2, 0);
    hipStreamAttachMemAsync(s2, nb_amax, 0);
    hipStreamAttachMemAsync(s2, nb_l, 0);
    hipStreamAttachMemAsync(s1, ridge_intercept, 0);

    rr_1<<<num_blocks, block_size_1d, 0, s1>>>(x, z, N, num_features);
    // dim3 num_blocks_2d(8, 8);
    // dim3 block_size_1d_2d(8, 8);
    // rr_1_0<<<num_blocks_2d, block_size_1d_2d, 0, s1>>>(x, r1_mean, r1_std, N, num_features);
    // rr_1_1<<<num_blocks_2d, block_size_1d_2d, 0, s1>>>(x, z, r1_mean, r1_std, N, num_features);

    nb_1<<<num_blocks, block_size_1d, 0, s2>>>(x, nb_feat_log_prob, r1, N, num_features, num_classes);

    rr_2<<<num_blocks, block_size_1d, 0, s1>>>(z, ridge_coeff, r2, N, num_features, num_classes);

    nb_2<<<num_blocks, block_size_1d, 0, s2>>>(r1, nb_amax, N, num_classes);

    nb_3<<<num_blocks, block_size_1d, 0, s2>>>(r1, nb_amax, nb_l, N, num_classes);

    rr_3<<<num_blocks, block_size_1d, 0, s1>>>(r2, ridge_intercept, N, num_classes);

    nb_4<<<num_blocks, block_size_1d, 0, s2>>>(r1, nb_l, N, num_classes);

    softmax<<<num_blocks, block_size_1d, 0, s2>>>(r1, N, num_classes);

    softmax<<<num_blocks, block_size_1d, 0, s1>>>(r2, N, num_classes);

    // Stream 1 waits stream 2;
    hipEvent_t e1;
    hipEventCreate(&e1);
    hipEventRecord(e1, s2);
    hipStreamWaitEvent(s1, e1, 0);

    argmax<<<num_blocks, block_size_1d, 0, s1>>>(r1, r2, r, N, num_classes);
    hipDeviceSynchronize();
}

void Benchmark6::execute_cudagraph(int iter) {
    if (iter == 0) {
        hipEvent_t ef;
        hipEventCreate(&ef);
        hipStreamBeginCapture(s1, hipStreamCaptureModeGlobal);
        hipEventRecord(ef, s1);
        hipStreamWaitEvent(s2, ef, 0);

        rr_1<<<num_blocks, block_size_1d, 0, s1>>>(x, z, N, num_features);
        // dim3 num_blocks_2d(8, 8);
        // dim3 block_size_1d_2d(8, 8);
        // rr_1_0<<<num_blocks_2d, block_size_1d_2d, 0, s1>>>(x, r1_mean, r1_std, N, num_features);
        // rr_1_1<<<num_blocks_2d, block_size_1d_2d, 0, s1>>>(x, z, r1_mean, r1_std, N, num_features);

        nb_1<<<num_blocks, block_size_1d, 0, s2>>>(x, nb_feat_log_prob, r1, N, num_features, num_classes);

        rr_2<<<num_blocks, block_size_1d, 0, s1>>>(z, ridge_coeff, r2, N, num_features, num_classes);

        nb_2<<<num_blocks, block_size_1d, 0, s2>>>(r1, nb_amax, N, num_classes);

        nb_3<<<num_blocks, block_size_1d, 0, s2>>>(r1, nb_amax, nb_l, N, num_classes);

        rr_3<<<num_blocks, block_size_1d, 0, s1>>>(r2, ridge_intercept, N, num_classes);

        nb_4<<<num_blocks, block_size_1d, 0, s2>>>(r1, nb_l, N, num_classes);

        softmax<<<num_blocks, block_size_1d, 0, s2>>>(r1, N, num_classes);

        softmax<<<num_blocks, block_size_1d, 0, s1>>>(r2, N, num_classes);

        // Stream 1 waits stream 2;
        hipEvent_t e1;
        hipEventCreate(&e1);
        hipEventRecord(e1, s2);
        hipStreamWaitEvent(s1, e1, 0);

        argmax<<<num_blocks, block_size_1d, 0, s1>>>(r1, r2, r, N, num_classes);

        hipStreamEndCapture(s1, &graph);
        hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    }
    hipGraphLaunch(graphExec, s1);
    err = hipStreamSynchronize(s1);
}

void Benchmark6::execute_cudagraph_manual(int iter) {
    if (iter == 0) {
        hipGraphCreate(&graph, 0);
        void* kernel_1_args[4] = {(void*)&x, (void*)&z, &N, &num_features};
        void* kernel_2_args[6] = {(void*)&x, (void*)&nb_feat_log_prob, (void*)&r1, &N, &num_features, &num_classes};
        void* kernel_3_args[6] = {(void*)&z, (void*)&ridge_coeff, (void*)&r2, &N, &num_features, &num_classes};
        void* kernel_4_args[4] = {(void*)&r1, (void*)&nb_amax, &N, &num_classes};
        void* kernel_5_args[5] = {(void*)&r1, (void*)&nb_amax, (void*)&nb_l, &N, &num_classes};
        void* kernel_6_args[5] = {(void*)&r2, (void*)&ridge_intercept, (void*)&nb_l, &N, &num_classes};
        void* kernel_7_args[4] = {(void*)&r1, (void*)&nb_l, &N, &num_classes};
        void* kernel_8_args[3] = {(void*)&r1, &N, &num_classes};
        void* kernel_9_args[3] = {(void*)&r2, &N, &num_classes};
        void* kernel_10_args[5] = {(void*)&r1, (void*)&r2, (void*)&r, &N, &num_classes};

        dim3 tb(block_size_1d);
        dim3 bs(num_blocks);

        add_node(kernel_1_args, kernel_1_params, (void*)rr_1, tb, bs, graph, &kernel_1, nodeDependencies);
        add_node(kernel_2_args, kernel_2_params, (void*)nb_1, tb, bs, graph, &kernel_2, nodeDependencies);

        nodeDependencies.push_back(kernel_1);
        add_node(kernel_3_args, kernel_3_params, (void*)rr_2, tb, bs, graph, &kernel_3, nodeDependencies);

        nodeDependencies.clear();
        nodeDependencies.push_back(kernel_2);
        add_node(kernel_4_args, kernel_4_params, (void*)nb_2, tb, bs, graph, &kernel_4, nodeDependencies);

        nodeDependencies.clear();
        nodeDependencies.push_back(kernel_4);
        add_node(kernel_5_args, kernel_5_params, (void*)nb_3, tb, bs, graph, &kernel_5, nodeDependencies);

        // nodeDependencies.clear();
        // nodeDependencies.push_back(kernel_3);
        // add_node(kernel_6_args, kernel_6_params, (void*)rr_3, tb, bs, graph, &kernel_6, nodeDependencies);

        // nodeDependencies.clear();
        // nodeDependencies.push_back(kernel_5);
        // add_node(kernel_7_args, kernel_7_params, (void*)nb_4, tb, bs, graph, &kernel_7, nodeDependencies);

        // nodeDependencies.clear();
        // nodeDependencies.push_back(kernel_7);
        // add_node(kernel_8_args, kernel_8_params, (void*)softmax, tb, bs, graph, &kernel_8, nodeDependencies);

        // nodeDependencies.clear();
        // nodeDependencies.push_back(kernel_6);
        // add_node(kernel_9_args, kernel_9_params, (void*)softmax, tb, bs, graph, &kernel_9, nodeDependencies);

        // nodeDependencies.clear();
        // nodeDependencies.push_back(kernel_8);
        // nodeDependencies.push_back(kernel_9);
        // add_node(kernel_10_args, kernel_10_params, (void*)argmax, tb, bs, graph, &kernel_10, nodeDependencies);

        // rr_1<<<num_blocks, block_size_1d, 0, s1>>>(x, z, N, num_features);
        // nb_1<<<num_blocks, block_size_1d, 0, s2>>>(x, nb_feat_log_prob, r1, N, num_features, num_classes);
        // rr_2<<<num_blocks, block_size_1d, 0, s1>>>(z, ridge_coeff, r2, N, num_features, num_classes);
        // nb_2<<<num_blocks, block_size_1d, 0, s2>>>(r1, nb_amax, N, num_classes);
        // nb_3<<<num_blocks, block_size_1d, 0, s2>>>(r1, nb_amax, nb_l, N, num_classes);
        // rr_3<<<num_blocks, block_size_1d, 0, s1>>>(r2, ridge_intercept, N, num_classes);
        // nb_4<<<num_blocks, block_size_1d, 0, s2>>>(r1, nb_l, N, num_classes);
        // softmax<<<num_blocks, block_size_1d, 0, s2>>>(r1, N, num_classes);
        // softmax<<<num_blocks, block_size_1d, 0, s1>>>(r2, N, num_classes)
        // argmax<<<num_blocks, block_size_1d, 0, s1>>>(r1, r2, r, N, num_classes);

        hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    }
    hipGraphLaunch(graphExec, s1);
    err = hipStreamSynchronize(s1);
}

std::string Benchmark6::print_result(bool short_form) {
    if (short_form) {
        return std::to_string(r[0]);
    } else {
        std::string res = "[";
        for (int j = 0; j < 10; j++) {
            res += std::to_string(r[j]) + ", ";
        }
        return res + ", ...]";
    }
}
