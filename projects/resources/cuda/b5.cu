#include "hip/hip_runtime.h"
#include "b5.cuh"

//////////////////////////////
//////////////////////////////

double R = 0.08;
double V = 0.3;
double T = 1.0;
double K = 60.0;

__device__ inline double
cndGPU(double d) {
    const double A1 = 0.31938153f;
    const double A2 = -0.356563782f;
    const double A3 = 1.781477937f;
    const double A4 = -1.821255978f;
    const double A5 = 1.330274429f;
    const double RSQRT2PI = 0.39894228040143267793994605993438f;

    double K = 1.0 / (1.0 + 0.2316419 * fabs(d));

    double cnd = RSQRT2PI * exp(-0.5f * d * d) *
                 (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

    if (d > 0)
        cnd = 1.0 - cnd;

    return cnd;
}

extern "C" __global__ void
bs(const double *x, double *y, int N, double R, double V, double T, double K) {
    double sqrtT = 1.0 / rsqrt(T);
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N;
         i += blockDim.x * gridDim.x) {
        double expRT;
        double d1, d2, CNDD1, CNDD2;
        d1 = (log(x[i] / K) + (R + 0.5 * V * V) * T) / (V * sqrtT);
        d2 = d1 - V * sqrtT;

        CNDD1 = cndGPU(d1);
        CNDD2 = cndGPU(d2);

        // Calculate Call and Put simultaneously
        expRT = exp(-R * T);
        y[i] = x[i] * CNDD1 - K * expRT * CNDD2;
    }
}

void prefetch(double *x, double *y, hipStream_t &s, int N) {
    int pascalGpu = 0;
    hipDeviceGetAttribute(&pascalGpu, hipDeviceAttribute_t::hipDeviceAttributeConcurrentManagedAccess, 0);
    if (pascalGpu) {
        hipMemPrefetchAsync(x, sizeof(double) * N, 0, s);
        hipMemPrefetchAsync(y, sizeof(double) * N, 0, s);
    }
}

//////////////////////////////
//////////////////////////////

void Benchmark5::alloc() {
    x = (double **)malloc(sizeof(double *) * M);
    y = (double **)malloc(sizeof(double *) * M);
    tmp_x = (double *)malloc(sizeof(double) * N);
    // hipHostRegister(tmp_x, sizeof(double) * N, 0);

    for (int i = 0; i < M; i++) {
        hipMallocManaged(&x[i], sizeof(double) * N);
        hipMallocManaged(&y[i], sizeof(double) * N);
    }
}

void Benchmark5::init() {
    for (int j = 0; j < N; j++) {
        tmp_x[j] = 60 - 0.5 + (double)rand() / RAND_MAX;
        for (int i = 0; i < M; i++) {
            x[i][j] = tmp_x[j];
            // y[i][j] = 0;
        }
    }

    s = (hipStream_t *)malloc(sizeof(hipStream_t) * M);
    for (int i = 0; i < M; i++) {
        err = hipStreamCreate(&s[i]);
    }
}

void Benchmark5::reset() {
    for (int i = 0; i < M; i++) {
        // memcpy(x[i], y, sizeof(int) * N);
        // hipMemcpy(x[i], y, sizeof(double) * N, hipMemcpyDefault);

        // hipMemcpyAsync(x[i], y, sizeof(int) * N, hipMemcpyHostToDevice,
        // s[i]);
        for (int j = 0; j < N; j++) {
            x[i][j] = tmp_x[j];
        }
    }
    // hipMemPrefetchAsync(x[0], sizeof(double) * N, 0, s[0]);
}

void Benchmark5::execute_sync(int iter) {
    for (int j = 0; j < M; j++) {
        bs<<<num_blocks, block_size_1d>>>(x[j], y[j], N, R, V, T, K);
        err = hipDeviceSynchronize();
    }
}

void Benchmark5::execute_async(int iter) {
    for (int j = 0; j < M; j++) {
        hipStreamAttachMemAsync(s[j], x[j], sizeof(double) * N);
        hipStreamAttachMemAsync(s[j], y[j], sizeof(double) * N);
        // if (j > 0) hipMemPrefetchAsync(y[j - 1], sizeof(double) * N, hipCpuDeviceId, s[j - 1]);
        bs<<<num_blocks, block_size_1d, 0, s[j]>>>(x[j], y[j], N, R, V, T, K);
        // if (j < M - 1) hipMemPrefetchAsync(x[j + 1], sizeof(double) * N, 0, s[j + 1]);
    }

    // Last tile;
    // hipMemPrefetchAsync(y[M - 1], sizeof(double) * N, hipCpuDeviceId, s[M - 1]);

    for (int j = 0; j < M; j++) {
        err = hipStreamSynchronize(s[j]);
    }
}

void Benchmark5::execute_cudagraph(int iter) {
    if (iter == 0) {
        for (int j = 0; j < M; j++) {
            hipStreamBeginCapture(s[j], hipStreamCaptureModeGlobal);
            prefetch(x[j], y[j], s[j], N);
            bs<<<num_blocks, block_size_1d, 0, s[j]>>>(x[j], y[j], N, R, V, T, K);
            hipStreamEndCapture(s[j], &graphs[j]);
            hipGraphInstantiate(&graphExec[j], graphs[j], NULL, NULL, 0);
        }
    }
    for (int j = 0; j < M; j++) {
        hipGraphLaunch(graphExec[j], s[j]);
    }
    for (int j = 0; j < M; j++) {
        hipStreamSynchronize(s[j]);
    }
}

void Benchmark5::execute_cudagraph_manual(int iter) {
    if (iter == 0) {
        hipGraphCreate(&graphs[0], 0);
        for (int j = 0; j < M; j++) {
            void *kernel_args[7] = {(void *)&x[j], (void *)&y[j], &N, &R, &V, &T, &K};

            dim3 tb(block_size_1d);
            dim3 b_size(num_blocks);

            // bs<<<num_blocks, block_size_1d, 0, s[j]>>>(x[j], y[j], N, R, V, T, K);
            add_node(kernel_args, kernel_params[j], (void *)bs, b_size, tb, graphs[0], &kernels[j], nodeDependencies);
        }
        hipGraphInstantiate(&graphExec[0], graphs[0], NULL, NULL, 0);
    }
    hipGraphLaunch(graphExec[0], s[0]);
    err = hipStreamSynchronize(s[0]);
}

void Benchmark5::execute_cudagraph_single(int iter) {
    if (iter == 0) {
        hipStreamBeginCapture(s[0], hipStreamCaptureModeGlobal);
        for (int j = 0; j < M; j++) {
            prefetch(x[j], y[j], s[0], N);
            bs<<<num_blocks, block_size_1d, 0, s[0]>>>(x[j], y[j], N, R, V, T, K);
        }
        hipStreamEndCapture(s[0], &graphs[0]);
        hipGraphInstantiate(&graphExec[0], graphs[0], NULL, NULL, 0);
    }
    hipGraphLaunch(graphExec[0], s[0]);
    hipStreamSynchronize(s[0]);
}

std::string
Benchmark5::print_result(bool short_form) {
    if (short_form) {
        return std::to_string(y[0][0]);
    } else {
        std::string res = "[";
        for (int j = 0; j < M; j++) {
            res += std::to_string(y[j][0]) + ", ";
        }
        return res + ", ...]";
    }
}
