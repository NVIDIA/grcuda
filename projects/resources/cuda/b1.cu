#include "hip/hip_runtime.h"
#include "b1.cuh"

//////////////////////////////
//////////////////////////////

__global__ void square(const float *x, float *y, int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        // float tmp = x[i];
        // float sum = 0;
        // for (int j = 0; j < 4; j++) {
        //     sum += tmp + j;
        // }

        y[i] = x[i] * x[i];  // tmp + tmp * tmp / 2 + tmp * tmp * tmp / 6;
    }
}

__inline__ __device__ float warp_reduce(float val) {
    int warp_size = 32;
    for (int offset = warp_size / 2; offset > 0; offset /= 2)
        val += __shfl_down_sync(0xFFFFFFFF, val, offset);
    return val;
}

// __device__ float atomicAddDouble(float* address, float val) {
//     unsigned long long int* address_as_ull = (unsigned long long int*) address;
//     unsigned long long int old = *address_as_ull, assumed;
//     do {
//         assumed = old;
//         old = atomicCAS(address_as_ull, assumed, __float_as_longlong(val + __longlong_as_float(assumed)));
//     } while (assumed != old);
//     return __longlong_as_float(old);
// }

__global__ void reduce(const float *x, const float *y, float *z, int N) {
    int warp_size = 32;
    float sum = float(0);
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        sum += x[i] - y[i];
    }
    sum = warp_reduce(sum);                    // Obtain the sum of values in the current warp;
    if ((threadIdx.x & (warp_size - 1)) == 0)  // Same as (threadIdx.x % warp_size) == 0 but faster
        atomicAdd(z, sum);                     // The first thread in the warp updates the output;
}

void Benchmark1::prefetch(hipStream_t &s1, hipStream_t &s2) {
    if (pascalGpu) {
        hipMemPrefetchAsync(x, sizeof(float) * N, 0, s1);
        hipMemPrefetchAsync(y, sizeof(float) * N, 0, s2);
        hipMemPrefetchAsync(x1, sizeof(float) * N, 0, s1);
        hipMemPrefetchAsync(y1, sizeof(float) * N, 0, s2);
    }
}

//////////////////////////////
//////////////////////////////

void Benchmark1::alloc() {
    err = hipMallocManaged(&x, sizeof(float) * N);
    err = hipMallocManaged(&y, sizeof(float) * N);
    err = hipMallocManaged(&x1, sizeof(float) * N);
    err = hipMallocManaged(&y1, sizeof(float) * N);
    err = hipMallocManaged(&res, sizeof(float));

    err = hipStreamCreate(&s1);
    err = hipStreamCreate(&s2);
}

void Benchmark1::init() {
    for (int i = 0; i < N; i++) {
        x[i] = 1.0 / (i + 1);
        y[i] = 2.0 / (i + 1);
    }
}

void Benchmark1::reset() {
    for (int i = 0; i < N; i++) {
        x[i] = 1.0 / (i + 1);
        y[i] = 2.0 / (i + 1);
    }
    res[0] = 0.0;
}

void Benchmark1::execute_sync(int iter) {
    if (do_prefetch && pascalGpu) {
        hipMemPrefetchAsync(x, sizeof(float) * N, 0, 0);
        hipMemPrefetchAsync(x1, sizeof(float) * N, 0, 0);
        hipMemPrefetchAsync(y, sizeof(float) * N, 0, 0);
        hipMemPrefetchAsync(y1, sizeof(float) * N, 0, 0);
        hipMemPrefetchAsync(res, sizeof(float), 0, 0);
    }

    square<<<num_blocks, block_size_1d>>>(x, x1, N);
    err = hipDeviceSynchronize();
    square<<<num_blocks, block_size_1d>>>(y, y1, N);
    err = hipDeviceSynchronize();
    reduce<<<num_blocks, block_size_1d>>>(x1, y1, res, N);
    err = hipDeviceSynchronize();
}

void Benchmark1::execute_async(int iter) {
    if (!pascalGpu || stream_attach) {
        hipStreamAttachMemAsync(s1, x, sizeof(float) * N);
        hipStreamAttachMemAsync(s1, x1, sizeof(float) * N);
        hipStreamAttachMemAsync(s2, y, sizeof(float) * N);
        hipStreamAttachMemAsync(s2, y1, sizeof(float) * N);
    }
    if (pascalGpu && do_prefetch) {
        hipMemPrefetchAsync(x, sizeof(float) * N, 0, s1);
        hipMemPrefetchAsync(x1, sizeof(float) * N, 0, s1);
        hipMemPrefetchAsync(y, sizeof(float) * N, 0, s2);
        hipMemPrefetchAsync(y1, sizeof(float) * N, 0, s2);
        hipMemPrefetchAsync(res, sizeof(float), 0, s1);
    }

    square<<<num_blocks, block_size_1d, 0, s1>>>(x, x1, N);
    square<<<num_blocks, block_size_1d, 0, s2>>>(y, y1, N);

    // Stream 1 waits stream 2;
    hipEvent_t e1;
    hipEventCreate(&e1);
    hipEventRecord(e1, s2);
    hipStreamWaitEvent(s1, e1, 0);

    reduce<<<num_blocks, block_size_1d, 0, s1>>>(x1, y1, res, N);
    hipStreamSynchronize(s1);
}

void Benchmark1::execute_cudagraph(int iter) {
    if (iter == 0) {
        hipEvent_t ef;
        hipEventCreate(&ef);
        hipStreamBeginCapture(s1, hipStreamCaptureModeGlobal);
        hipEventRecord(ef, s1);
        hipStreamWaitEvent(s2, ef, 0);

        // prefetch(s1, s2);

        square<<<num_blocks, block_size_1d, 0, s1>>>(x, x1, N);
        square<<<num_blocks, block_size_1d, 0, s2>>>(y, y1, N);
        // Stream 1 waits stream 2;
        hipEvent_t e1;
        hipEventCreate(&e1);
        hipEventRecord(e1, s2);
        hipStreamWaitEvent(s1, e1, 0);
        reduce<<<num_blocks, block_size_1d, 0, s1>>>(x1, y1, res, N);

        hipStreamEndCapture(s1, &graph);
        hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    }
    hipGraphLaunch(graphExec, s1);
    err = hipStreamSynchronize(s1);
}

void Benchmark1::execute_cudagraph_manual(int iter) {
    if (iter == 0) {
        hipGraphCreate(&graph, 0);
        void *kernel_1_args[3] = {(void *)&x, (void *)&x1, &N};
        void *kernel_2_args[3] = {(void *)&y, (void *)&y1, &N};
        void *kernel_3_args[4] = {(void *)&x1, (void *)&y1, (void *)&res, &N};

        dim3 tb(block_size_1d);
        dim3 bs(num_blocks);

        // square<<<num_blocks, block_size_1d, 0, s1>>>(x, x1, N);
        add_node(kernel_1_args, kernel_1_params, (void *)square, bs, tb, graph, &kernel_1, nodeDependencies);

        // square<<<num_blocks, block_size_1d, 0, s2>>>(y, y1, N);
        add_node(kernel_2_args, kernel_2_params, (void *)square, bs, tb, graph, &kernel_2, nodeDependencies);

        // reduce<<<num_blocks, block_size_1d, 0, s1>>>(x1, y1, res, N);
        nodeDependencies.push_back(kernel_1);
        nodeDependencies.push_back(kernel_2);
        add_node(kernel_3_args, kernel_3_params, (void *)reduce, bs, tb, graph, &kernel_3, nodeDependencies);

        hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    }
    hipGraphLaunch(graphExec, s1);
    err = hipStreamSynchronize(s1);
}

void Benchmark1::execute_cudagraph_single(int iter) {
    if (iter == 0) {
        hipStreamBeginCapture(s1, hipStreamCaptureModeGlobal);

        // prefetch(s1, s1);

        square<<<num_blocks, block_size_1d, 0, s1>>>(x, x1, N);
        square<<<num_blocks, block_size_1d, 0, s1>>>(y, y1, N);
        reduce<<<num_blocks, block_size_1d, 0, s1>>>(x1, y1, res, N);

        hipStreamEndCapture(s1, &graph);
        hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    }
    hipGraphLaunch(graphExec, s1);
    err = hipStreamSynchronize(s1);
}

std::string Benchmark1::print_result(bool short_form) {
    return std::to_string(res[0]);
}
