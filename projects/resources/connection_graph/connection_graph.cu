#include <iostream>
#include <stdio.h>
#include <nvml.h>
#include <hip/hip_runtime.h>
#include <iomanip>
#include <fstream>

#define N 500000000 // 500 MB

// Test bandwidth between two GPUs;
float dtod_copy(size_t size, int from, int to) {
	int *pointers[2];

	hipSetDevice(from);
	hipDeviceEnablePeerAccess(to, 0);
	hipMalloc(&pointers[0], size);

	hipSetDevice(to);
	hipDeviceEnablePeerAccess(from, 0);
	hipMalloc(&pointers[1], size);

	hipEvent_t begin, end;
	hipEventCreate(&begin);
	hipEventCreate(&end);

	hipEventRecord(begin);
	hipMemcpyAsync(pointers[0], pointers[1], size, hipMemcpyDeviceToDevice);
	hipEventRecord(end);
	hipEventSynchronize(end);

	float elapsed;
	hipEventElapsedTime(&elapsed, begin, end);
	elapsed /= 1000;

	hipSetDevice(from);
	hipFree(pointers[0]);

	hipSetDevice(to);
	hipFree(pointers[1]);

	hipEventDestroy(end);
	hipEventDestroy(begin);
	hipSetDevice(from);

	return elapsed;
}

// Test bandwidth from the CPU to a device;
float htod_copy(size_t size, int device_id) {
	int *pointer, *d_pointer;

	hipSetDevice(device_id);
	hipMalloc(&d_pointer, size);
	hipHostMalloc(&pointer, size, hipHostMallocDefault);

	hipEvent_t begin, end;
	hipEventCreate(&begin);
	hipEventCreate(&end);

	hipEventRecord(begin);
	hipMemcpyAsync(d_pointer, pointer, size, hipMemcpyHostToDevice);
	hipEventRecord(end);
	hipEventSynchronize(end);

	float elapsed;
	hipEventElapsedTime(&elapsed, begin, end);
	elapsed /= 1000;

	hipSetDevice(device_id);
	hipFree(d_pointer);

	hipEventDestroy(end);
	hipEventDestroy(begin);

	return elapsed;
}

int main() {
    int gpu_number = 0;

	hipGetDeviceCount(&gpu_number);  
	printf("number of devices = %d\n", gpu_number);

	double **bandwidths = (double**) malloc(gpu_number * sizeof(double*));
	for (int i = 0; i < gpu_number; i++) {
		bandwidths[i] = (double*) malloc(gpu_number * sizeof(double));
    }
	std::ofstream out_file;
	// This is not safe, I guess;
    std::string grcuda_home = getenv("GRCUDA_HOME");
	out_file.open(grcuda_home + "/projects/resources/connection_graph/datasets/connection_graph.csv");
	out_file << "From,To,Bandwidth\n";

	for (int i = 0; i < gpu_number; i++) {
        // Measure CPU-to-GPU transfer time;
		double time_htod = htod_copy(N, 1);
		printf("\nfrom: Host, to: %d, time spent: %f, transfer rate: %f GB/s \n",i, time_htod, (float(N) / 1000000000.0) / time_htod);
		out_file << std::setprecision(15) << "-1" << "," << i << "," << (double(N) /1000000000.0) / time_htod << "\n";
		
        for (int j = 0 ; j < gpu_number; j++) {
            // Measure GPU-to-GPU transfer time;
			double time_dtod = dtod_copy(N, i, j);
			bandwidths[i][j] = (double(N) / 1000000000.0) / time_dtod;
			printf("from: %d, to: %d, time spent: %f, transfer rate: %f GB/s \n", i, j, time_dtod, bandwidths[i][j]);
			out_file << i << "," << j << "," << bandwidths[i][j] << "\n";
		}
	}
	out_file.close();
	return 0;
}
